#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>

#define THREADS_PER_BLOCK 512

__global__ void innerProd(float *aa, float *bb, float *cc)
{
   __shared__ float temp[THREADS_PER_BLOCK];
   int index = threadIdx.x + blockIdx.x* blockDim.x;
   temp[threadIdx.x] = aa[index]*bb[index];

   *cc = 0; // Initialized to avoid memory problems. See comments
            // below, next to the free and cudaFree commands.

   // No thread goes beyond this point until all of them
   // have reached it. Threads are only synchronized within
   // a block.
   __syncthreads();

   //  Thread 0 sums the pairwise products
   if (threadIdx.x == 0) {
     float sum = 0;
     for (int i = 0; i < THREADS_PER_BLOCK; i++){
       sum += temp[i];
     }
      // Use atomicAdd to avoid different blocks accessing cc at the
      // same time (race condition). The atomic opperation enables
      // read-modify-write to be performed by a block without interruption.
      //*cc += sum;
    atomicAdd(cc, sum);
   }

}


void cuda_function(float *d_a, float *d_b, float *d_c, float *a, float *b, float *c, int NN)
{

  #define NUMBER_OF_BLOCKS (NN/THREADS_PER_BLOCK)
  float GPU_profile;
  float size = NN * sizeof(float);

  // ----- Variables to profile the execution time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
	
  // In the GPU ------------------------------------------
  hipMalloc((void**)&d_a, size);
  hipMalloc((void**)&d_b, size);
  hipMalloc((void**)&d_c, sizeof(float));

  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

  // Call kernel
  hipEventRecord(start);
  innerProd<<<NUMBER_OF_BLOCKS, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);
  hipEventRecord(stop);

  hipMemcpy(c, d_c, sizeof(float), hipMemcpyDeviceToHost);
  hipEventSynchronize(stop);

  // Elapsed time -- GPU
  hipEventElapsedTime(&GPU_profile, start, stop);
  // -----------------------------------------------------

  std::cout << "NUMBER_OF_BLOCKS = " << NUMBER_OF_BLOCKS << "\n";
  std::cout << "c = " << *c << "\n";
  std::cout << "Kernel execution time in GPU = " << GPU_profile <<
        " milliseconds" << "\n";

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
